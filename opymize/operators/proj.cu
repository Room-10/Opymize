
#include <hip/hip_runtime.h>

#ifdef L1_NORMS_PROJ
__global__ void l1normsproj(TYPE_T *x)
{
    /* This function makes heavy use of registers (34 32-bit registers), so
     * that it will not run with more than 960 threads per block on compute
     * capability 2.x!
     *
     * x_i = proj(x_i, lbd)
     */
#if (M1 <= M2)
// A := x_i, a (M1 x M2)-matrix
#define LIM M2
#define STEP1 M1
#define STEP2 (1)
#else
// A := x_i^T, a (M2 x M1)-matrix
#define LIM M1
#define STEP1 (1)
#define STEP2 M1
#endif

    // global thread index
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    // stay inside maximum dimensions
    if (i >= N) return;

    // iteration variables and misc.
    int mm;
    TYPE_T *xi = &x[i*(M1*M2)];
    TYPE_T norm = 0.0;

#if (M1 == 1 || M2 == 1 || matrixnorm == 'F')
    for (mm = 0; mm < M1*M2; mm++) {
        norm += xi[mm]*xi[mm];
    }

    if (norm > lbd*lbd) {
        norm = lbd/SQRT(norm);
        for (mm = 0; mm < M1*M2; mm++) {
            xi[mm] *= norm;
        }
    }
#elif (M1 == 2 || M2 == 2)
    TYPE_T C11 = 0.0, C12 = 0.0, C22 = 0.0,
           V11 = 0.0, V12 = 0.0, V21 = 0.0, V22 = 0.0,
           M11 = 0.0, M12 = 0.0, M21 = 0.0, M22 = 0.0,
           s1 = 0.0, s2 = 0.0,
           trace, d, lmax, lmin, smax, smin;

    // C = A^T A, a (2 x 2)-matrix
    for (mm = 0; mm < LIM; mm++) {
        C11 += xi[mm*STEP1 + 0*STEP2]*xi[mm*STEP1 + 0*STEP2];
        C12 += xi[mm*STEP1 + 0*STEP2]*xi[mm*STEP1 + 1*STEP2];
        C22 += xi[mm*STEP1 + 1*STEP2]*xi[mm*STEP1 + 1*STEP2];
    }

    // Compute eigenvalues
    trace = C11 + C22;
    d = SQRT(FMAX(0.0, 0.25*trace*trace - (C11*C22 - C12*C12)));
    lmax = FMAX(0.0, 0.5*trace + d);
    lmin = FMAX(0.0, 0.5*trace - d);
    smax = SQRT(lmax);
    smin = SQRT(lmin);

    if (smax > lbd) {
        // Compute orthonormal eigenvectors
        if (C12 == 0.0) {
            if (C11 >= C22) {
                V11 = 1.0; V12 = 0.0;
                V21 = 0.0; V22 = 1.0;
            } else {
                V11 = 0.0; V12 = 1.0;
                V21 = 1.0; V22 = 0.0;
            }
        } else {
            V11 = C12       ; V12 = C12;
            V21 = lmax - C11; V22 = lmin - C11;
            norm = HYPOT(V11, V21);
            V11 /= norm; V21 /= norm;
            norm = HYPOT(V12, V22);
            V12 /= norm; V22 /= norm;
        }

        // Thresholding of eigenvalues
        s1 = FMIN(smax, lbd)/smax;
        s2 = FMIN(smin, lbd);
        s2 = (smin > 0.0) ? s2/smin : 0.0;

        // M = V * diag(s) * V^T
        M11 = s1*V11*V11 + s2*V12*V12;
        M12 = s1*V11*V21 + s2*V12*V22;
        M21 = s1*V21*V11 + s2*V22*V12;
        M22 = s1*V21*V21 + s2*V22*V22;

        // proj(A) = A * M
        for (mm = 0; mm < LIM; mm++) {
            // s1, s2 now used as temp. variables
            s1 = xi[mm*STEP1 + 0*STEP2];
            s2 = xi[mm*STEP1 + 1*STEP2];
            xi[mm*STEP1 + 0*STEP2] = s1*M11 + s2*M21;
            xi[mm*STEP1 + 1*STEP2] = s1*M12 + s2*M22;
        }
    }
#endif
}
#endif

#ifdef EPIGRAPH_PROJ
#include <stdio.h>

inline __device__ void proj_plane(TYPE_T *a, TYPE_T *g)
{
    /* Compute the normal projection of g onto the subspace which
     * is orthogonal to (a,-1).
     *
     * This is equivalent to solving
     *
     *      minimize  0.5*<p,p> - <g,p>
     *          s.t.  a[0]*p[0] + a[1]*p[1] = p[2].
     *
     * The result is stored in g.
     */

    // fac : <(a,-1),g> / <(a,-1),(a,-1)>
    TYPE_T fac  = a[0]*g[0] + a[1]*g[1] + (-1)*g[2];
           fac /= a[0]*a[0] + a[1]*a[1] + (-1)*(-1);

    // g -= fac*(a,-1)
    g[0] -= fac*a[0];
    g[1] -= fac*a[1];
    g[2] -= fac*(-1);
}

inline __device__ void proj_line(TYPE_T *a0, TYPE_T *a1, TYPE_T *g)
{
    /* Compute the normal projection of g onto the 1-dimensional subspace which
     * is orthogonal to span{(a0,-1),(a1,-1)}.
     *
     * This is equivalent to solving
     *
     *      minimize  0.5*<p,p> - <g,p>
     *          s.t.  a0[0]*p[0] + a0[1]*p[1] = p[2],
     *                a1[0]*p[0] + a1[1]*p[1] = p[2].
     *
     * The result is stored in g.
     */

    // v : cross product of (a0,-1) and (a1,-1)
    TYPE_T v[3];
    v[0] = a0[1]*(-1)  -  (-1)*a1[1];
    v[1] =  (-1)*a1[0] - a0[0]*(-1) ;
    v[2] = a0[0]*a1[1] - a0[1]*a1[0];

    // fac : <g,v>/<v,v>
    TYPE_T fac  = v[0]*g[0] + v[1]*g[1] + v[2]*g[2];
           fac /= v[0]*v[0] + v[1]*v[1] + v[2]*v[2];

    // g = fac*v
    g[0] = fac*v[0];
    g[1] = fac*v[1];
    g[2] = fac*v[2];
}

inline __device__ void base_trafo_2d(TYPE_T *a0, TYPE_T *a1, TYPE_T *g)
{
    /* Express g in terms of {(a0,-1), (a1,-1)}.
     *
     * The result is stored in g so that
     *
     *      g[0]*(a0,-1) + g[1]*(a1,-1) = input
     */

    TYPE_T diff0 = a0[0] - a1[0];
    TYPE_T diff1 = a0[1] - a1[1];

    if (FABS(diff1) > FABS(diff0)) {
        g[0] = (g[1] + g[2]*a1[1])/diff1;
    } else {
        g[0] = (g[0] + g[2]*a1[0])/diff0;
    }

    // make use of -mu[0]-mu[1] = g[2]
    g[1] = -g[2] - g[0];
}

inline __device__ bool solve_2x2(TYPE_T *A, TYPE_T *b)
{
    /* Solve a 2x2 linear system of equations.
     *
     * If singular, nothing is written and `false` is returned (else `true`).
     *
     * The result is stored in b.
     */

    TYPE_T detA = A[0]*A[3] - A[1]*A[2];
    TYPE_T res0, res1;
    int row0 = 0;
    int row1 = 1;

    if (FABS(detA) < 1e-9) {
        printf("Warning: Singular matrix in solve_2x2, det(A)=%g\n", detA);
        return false;
    } else {
        if(FABS(A[row0*2 + 0]) < FABS(A[row1*2 + 0])) {
            // swap rows for numerical stability
            row0 = 1; row1 = 0;
        }
        res1 = A[row1*2 + 0]/A[row0*2 + 0];
        res0 = A[row1*2 + 1] - A[row0*2 + 1]*res1;
        res1 = (b[row1] - b[row0]*res1)/res0;
        res0 = (b[row0] - A[row0*2 + 1]*res1)/A[row0*2 + 0];
        b[0] = res0;
        b[1] = res1;
        return true;
    }
}

inline __device__ void base_trafo_3d(TYPE_T *a0, TYPE_T *a1, TYPE_T *a2, TYPE_T *g)
{
    /* Express g in terms of {(a0,-1), (a1,-1), (a2,-1)}.
     *
     * The result is stored in g so that
     *
     *      g[0]*(a0,-1) + g[1]*(a1,-1) + g[2]*(a2,-1) = input
     */

    TYPE_T matrix[4];
    matrix[0] = a0[0] - a2[0];
    matrix[1] = a1[0] - a2[0];
    matrix[2] = a0[1] - a2[1];
    matrix[3] = a1[1] - a2[1];
    g[0] = g[0] + g[2]*a2[0];
    g[1] = g[1] + g[2]*a2[1];
    solve_2x2(matrix, g);

    // make use of -mu[0]-mu[1]-mu[2] = g[2]
    g[2] = -g[2] - g[1] - g[0];
}

inline __device__ int array_index_of(int *array, int array_size, int val) {
    for (int i = 0; i < array_size; i++) {
        if (array[i] == val) {
            return i;
        }
    }
    return -1;
}

inline __device__ int array_argmin(TYPE_T *array, TYPE_T array_size) {
    TYPE_T min = array[0];
    TYPE_T argmin = 0;
    for (int i = 1; i < array_size; i++) {
        if (array[i] < min) {
            min = array[i];
            argmin = i;
        }
    }
    return argmin;
}

inline __device__ void solve_qp(TYPE_T *x, TYPE_T **A, TYPE_T *b, int N, TYPE_T *sol)
{
    /* This function solves
     *
     *      minimize  0.5*||y - x||**2   s.t.  A y <= b,
     *
     * using an active set method that assumes that at most three constraints
     * are active at the same time.
     *
     * For more details see Algorithm 16.3 in
     *
     *      Nocedal, Wright: Numerical Optimization (2nd Ed.). Springer, 2006.
     *
     *
     *  Args:
     *      x : shape (3,)
     *      A : shape (N,2); the matrix A is actually of shape (N,3), but the
     *          last column is not stored in memory because it has the constant
     *          value -1
     *      b : shape (N,)
     *      N : number of inequality constraints
     *      sol : shape (3,), the result is stored in `sol`
     */

    // iteration variables
    int k, l, _iter;

    // dir : search direction
    TYPE_T dir[3];
    TYPE_T step_min, step, Ax, Ad;

    // active and working set
    int active_set[3];
    int active_set_size = 0;
    int blocking = -1;

    // lagrange multipliers
    TYPE_T lambda[3];
    int lambda_argmin;

    // initialize with input
    for (k = 0; k < 3; k++) {
        sol[k] = x[k];
    }

    // determine initial feasible guess by increasing sol[2] if necessary
    for (l = 0; l < N; l++) {
        Ax = A[l][0]*sol[0] + A[l][1]*sol[1] - sol[2];
        if (Ax > b[l]) {
            sol[2] += Ax - b[l];
            active_set[0] = l;
            active_set_size = 1;
        }
    }

    // projections are idempotent (feasible inputs are mapped to themselves)
    if (active_set_size == 0) return;

    for (_iter = 0; _iter < term_maxiter; _iter++) {
        blocking = -1;

        // explicitely solve equality constrained helper QPs
        for (k = 0; k < 3; k++) {
            dir[k] = x[k] - sol[k];
        }

        if (active_set_size == 1) {
            proj_plane(A[active_set[0]], dir);
        } else if (active_set_size == 2) {
            proj_line(A[active_set[0]], A[active_set[1]], dir);
        }

        if (FABS(dir[0]) + FABS(dir[1]) + FABS(dir[2]) > 0) {
            // determine smallest step size at which a new (blocking)
            // constraint enters the active set
            step_min = 1.0;

            // iterate over constraints not in active set
            for (k = 0; k < N; k++) {
                if (-1 != array_index_of(active_set, active_set_size, k)) {
                    continue;
                }

                Ax = A[k][0]*sol[0]   + A[k][1]*sol[1]   - sol[2];
                Ad = A[k][0]*dir[0] + A[k][1]*dir[1] - dir[2];

                // dir is orthogonal to a0 and a1. However, by the following
                // check, dir can't be orthogonal to a blocking constraint,
                // hence (a0,a1,a2) is always linearly independent.
                if (Ad > term_tolerance) {
                    step = (b[k] - Ax)/Ad;
                    if (step < step_min && step > -term_tolerance) {
                        step_min = step;
                        blocking = k;
                    }
                }
            }

            // advance
            for (k = 0; k < 3; k++) {
                sol[k] += step_min*dir[k];
            }
        }

        if (blocking != -1) {
            // add blocking constraint to active set
            active_set[active_set_size++] = blocking;
        } else if (active_set_size == 1) {
            // no blocking constraint and only one active constraint means
            // we are at the exact orthogonal projection inside of a facet or
            // all blocking constraints were sorted out via Lagrange multipliers
            break;
        }

        if (active_set_size == 3 || blocking == -1) {
            // compute Lagrange multipliers lambda
            lambda[0] = x[0] - sol[0];
            lambda[1] = x[1] - sol[1];
            lambda[2] = x[2] - sol[2];

            if (active_set_size == 2) {
                // No blocking constraint: sol is exact orthogonal projection of
                // x onto orth{a0,a1}. Hence, x-sol is in span{a0,a1}.
                base_trafo_2d(A[active_set[0]], A[active_set[1]], lambda);
            } else if (active_set_size == 3) {
                // dir != 0 and a blocking constraint a2. In this case,
                // (a0,a1,a2) is linearly independent (see comment above).
                base_trafo_3d(A[active_set[0]], A[active_set[1]],
                              A[active_set[2]], lambda);
            }

            lambda_argmin = array_argmin(lambda, active_set_size);
            if (lambda[lambda_argmin] >= 0.0) {
                // KKT conditions of full problem are satisfied
                break;
            } else {
                // remove most negative lambda from active set
                active_set[lambda_argmin] = active_set[--active_set_size];
            }
        }
    }

    if (_iter == term_maxiter) {
        printf("Warning: active set method didn't converge within %d "
               "iterations.\n", term_maxiter);
    }

#if 0
    // check feasibility of result
    for (l = 0; l < N; l++) {
        Ax = A[l][0]*sol[0] + A[l][1]*sol[1] - sol[2];
        if (Ax - b[l] > 1e-3) {
            printf("Warning: solution is not primal feasible: "
                   "diff=%g.\n", Ax - b[l]);
            break;
        }
    }
#endif
}

__global__ void epigraphproj(TYPE_T *x)
{
    /* This function solves, for fixed j and i,
     *
     *      minimize  0.5*||y - x[j,i]||**2
     *          s.t.  A[i,j] y <= b[i,j],
     *
     * using an active set method. The result is stored in x[j,i].
     */

    // global thread index
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;

    // stay inside maximum dimensions
    if (j >= nregions || i >= nfuns) return;

    int k, l;
    TYPE_T result[3];
    TYPE_T *xji = &x[(j*nfuns + i)*3];
    TYPE_T *Aij[nsubpoints];
    TYPE_T bij[nsubpoints];

    // set up constraints
    int Nij = 0;
    for (k = 0; k < nsubpoints; k++) {
        l = J[j*nsubpoints + k];
        if (I[i*npoints + l]) {
            Aij[Nij] = &A_STORE[l*2];
            bij[Nij++] = B_STORE[i*npoints + l];
        }
    }

    // solve and write result to input array
    solve_qp(xji, Aij, bij, Nij, result);
    for (k = 0; k < 3; k++) {
        xji[k] = result[k];
    }
}
#endif
